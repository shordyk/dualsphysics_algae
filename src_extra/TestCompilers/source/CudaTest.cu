#include "hip/hip_runtime.h"
/*
 <DUALSPHYSICS>  Copyright (c) 2022 by Dr Jose M. Dominguez et al. (see http://dual.sphysics.org/index.php/developers/). 

 EPHYSLAB Environmental Physics Laboratory, Universidade de Vigo, Ourense, Spain.
 School of Mechanical, Aerospace and Civil Engineering, University of Manchester, Manchester, U.K.

 This file is part of DualSPHysics. 

 DualSPHysics is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser General Public License 
 as published by the Free Software Foundation; either version 2.1 of the License, or (at your option) any later version.
 
 DualSPHysics is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more details. 

 You should have received a copy of the GNU Lesser General Public License along with DualSPHysics. If not, see <http://www.gnu.org/licenses/>. 
*/

/// \file CudaTest.cu \brief Implements functions and CUDA kernels for simple test.

#include "CudaTest.h"

namespace cutest{

//==============================================================================
// Returns gridsize according parameters using X dimension.
//==============================================================================
inline dim3 GetSimpleGridSize(unsigned n,unsigned blocksize){
  const unsigned nb=unsigned(n+blocksize-1)/blocksize;//-Total number of blocks.
  return(dim3(nb,1,1));
}

//------------------------------------------------------------------------------
/// Computes r=v^2.
//------------------------------------------------------------------------------
__global__ void KerPow2(int np,const float *v,float *r)
{
  int cp=blockIdx.x*blockDim.x + threadIdx.x;
  if(cp<np)r[cp]=v[cp]*v[cp];
}

//==============================================================================
/// Call cuda kenel to compute r=v^2.
//==============================================================================
void Pow2(int n,const float *v,float *r){
  if(n){
    const unsigned bksize=256;
    dim3 sgrid=GetSimpleGridSize(n,bksize);
    KerPow2 <<<sgrid,bksize>>> (n,v,r);
  }
}

}


